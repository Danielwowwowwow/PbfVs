#include "hip/hip_runtime.h"
#include "../include/boundary_gpu.h"

#include "../include/shared_math.h"
#include "../include/cuda_basic.h"

namespace pbf {
namespace impl_ {
    __global__ static void ApplyBoundaryConstraintKernel(const float3 boundary_pos,
        const float3 boundary_vel, const float3 boundary_normal, const int num_ptcs,
        float3* positions, float3* velocities) 
    {
        // precondition: |boundary_normal| is normalized, length is 1. 
		const int ptc_i = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (ptc_i >= num_ptcs) return;

        float3 pos = positions[ptc_i];
        float3 vel = velocities[ptc_i];
        const float plane_to_ptc_dist = dot(pos - boundary_pos, boundary_normal);
        if (plane_to_ptc_dist < kFloatEpsilon) { 
            // particle is out side the boundary
            const float3 proj_pos = pos - (plane_to_ptc_dist * boundary_normal);
            pos = proj_pos;

            // make sure |boundary_vel| is large enough to make the computation stable.
            const float3 proj_vel = dot(vel, boundary_vel) * boundary_vel / dot(boundary_vel, boundary_vel);
            // original particle velocity can be decompoosed into two components:
            // projected vel and perpendicular vel.
            const float3 proj_vel_diff = proj_vel - boundary_vel;
            if (dot(proj_vel_diff, boundary_normal) < kFloatEpsilon) {
                // particle projected velocity should be the same as that of the boundary
                const float3 perp_vel = vel - proj_vel;
                vel = boundary_vel + perp_vel;
            }
        }
        positions[ptc_i] = pos;
        velocities[ptc_i] = vel;
    }
} // namespace impl_

    void BoundaryConstraintGpu::SetPsAdaptor(std::shared_ptr<ParticleSystemGpuAdaptor> pa) {
        ps_adaptor_ = pa;
    }
    
    void BoundaryConstraintGpu::ApplyAtBoundary_(const BoundaryPlane& bp) {
        const float3 boundary_pos = Convert(bp.position);
        const float3 boundary_vel = Convert(bp.velocity);
        const float3 boundary_normal = Convert(bp.normal);

        const int num_ptcs = ps_adaptor_->NumParticles();
        const int num_blocks_ptc = ((num_ptcs + kNumThreadPerBlock - 1) / kNumThreadPerBlock);

        impl_::ApplyBoundaryConstraintKernel<<<num_blocks_ptc, kNumThreadPerBlock>>>(boundary_pos,
            boundary_vel, boundary_normal, num_ptcs, 
            ps_adaptor_->PositionsPtr(), ps_adaptor_->VelocitiesPtr());
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
    }
} // namespace pbf