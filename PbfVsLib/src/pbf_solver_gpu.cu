#include "hip/hip_runtime.h"
#include "../include/pbf_solver_gpu.h"

// CUDA
#include "hip/hip_runtime.h"
#include <thrust\device_vector.h>

namespace pbf {
namespace impl_ {
	// ParticleSystemGpu
	//
	// CellGridGpu (equivalent to SpatialHash on CPU)
	// - Need to config the grid size and cell size before usage. Once
	//   configured, they are not allowed to be modified. Not adaptive.
	// - active cell: a cell that contains at least one particle.
	//
	// Example: a particle system of 8 particles, a cell grid of 5 cells.
	// We will illustrate the necessary arrays for updating the cell grid.
	//
	// | 0 | 1 | 2 | 3 | 4 |  cell index 
	// cell_num_ptcs
	//   3   0   1   4   0
	// - size: #cells
	// - number of particles in each cell, including inactive(empty) ones
	// - sum of this array is the total number of particles
	//
	// cell_is_active_flags
	//   1   0   1   1   0
	// - size: #cells
	// - sum of this array is the number of active cells
	//
	// cell_to_active_cell_indices
	//   0   1   1   2   3
	// - size: #cells
	// - a prefix scan of |cell_is_active_flags|
	//
	// active_cell_num_ptcs
	//   3   1   4
	// - size: #active cells
	// - a compact of |cell_num_ptcs| accoording to |cell_to_active_cell_indices|
	// - sum of this array is the total number of particles
	//
	// ptc_begins_in_active_cell 
	//   0   3   4
	// - size: #active cells
	// - beginning index of the particle in each (active) cell in 
	//   |cell_ptc_indices|
	// - a prefix scan of |active_cell_num_ptcs|
	//
	// cell_ptc_indices
	// - size: #ptcs
	// - each slot stores a particle index in the particle system. these particle
	//   indices are arranged in a way that particles within the same grid cell
	//   are continuously stored inside |cell_ptc_indices|.
	//
	// ptc_offsets_within_cell
	// - size: #particles
	// - for any given particle index, |p_i|, we can get its position, |pos_i|,
	//   and its cell, |cell_i|. Then:
	//	 // the active cell index of |cell_i| in which |ptc_i| lives
	//	 ac_idx = cell_to_active_cell_indices[cell_i];
	//   // the beginning index of the particles within |cell_i|
	//	 // in |cell_ptc_indices|
	//   ptc_begin_idx = ptc_begins_in_active_cell[ac_index];
	//   p_i' = ptc_begin_idx + ptc_offset_within_cell[p_i]; 
	//   assert(p_i == p_i');
	//
	// Find neighbors for each particle:
	// ptc_num_neighbors
	// - size: #particles
	// - stores the number of neighbor particles for each particle
	//
	// ptc_neighbor_begins
	// - size: #particles
	// - ~[p_i] stores the beginning index of particle |p_i|'s neighbors
	//   in |ptc_neighbor_indices|
	//
	// ptc_neighbor_indices
	// - size: sum of |ptc_num_neighbors| 
	
	__device__ int3 GetCell(float3 pos, float cell_sz) {
		const float cell_sz_recpr = 1.0f / cell_sz;
		int cx = (int)(pos.x * cell_sz_recpr);
		int cy = (int)(pos.y * cell_sz_recpr);
		int cz = (int)(pos.z * cell_sz_recpr);
		return make_int3(cx, cy, cz);
	}

	__device__ int GetCellIndex(int3 cell, int3 num_cells_dim) {
		int result = cell.y * num_cells_dim.z;
		result = (result + cell.z) * num_cells_dim.x;
		result += cell.x;
		return result;
	}

	__device__ bool IsValidCell(int3 cell, int3 num_cells_dim) {
		return ((0 <= cell.x && cell.x < num_cells_dim.x) &&
			(0 <= cell.y && cell.y < num_cells_dim.y) &&
			(0 <= cell.z && cell.z < num_cells_dim.z));
	}

	void ResetNumPtcsInCell(thrust::device_vector<int>* cell_num_ptcs) {
		const size_t sz = cell_num_ptcs->size();
		cell_num_ptcs->assign(sz, 0);
	}

	__global__ void CountNumPtcsInCell(const float3* positions, 
		const int num_ptcs, const float cell_sz,
		const int3 num_cells_dim, int* cell_num_ptcs,
		int* ptc_offset_within_cell) {
		const int ptc_i = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (ptc_i >= num_ptcs) return;
		int3 ptc_cell = GetCell(positions[ptc_i], cell_sz);
		int cell_index = GetCellIndex(ptc_cell, num_cells_dim);
		int offs = atomicAdd(&cell_num_ptcs[cell_index], 1);
		ptc_offset_within_cell[ptc_i] = offs;
	}

} // namespace impl_
} // namespace pbf