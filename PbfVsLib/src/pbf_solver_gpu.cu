#include "hip/hip_runtime.h"
#include "..\include\pbf_solver_gpu.h"

// CUDA
#include "..\include\cuda_basic.h"
#include "..\include\hip/hip_vector_types.h"
#include <thrust\scan.h>
#include <thrust\execution_policy.h>
#include <thrust\copy.h>

namespace pbf {

float3 Convert(const point_t& pt) {
	return make_float3(pt.x, pt.y, pt.z);
}

namespace impl_ {
	constexpr int kNumThreadPerBlock = 256;
	
	// ParticleSystemGpu
	//
	// CellGridGpu (equivalent to SpatialHash on CPU)
	// - Need to config the grid size and cell size before usage. Once
	//   configured, they are not allowed to be modified. Not adaptive.
	// - active cell: a cell that contains at least one particle.
	//
	// Example: a particle system of 8 particles, a cell grid of 5 cells.
	// We will illustrate the necessary arrays for updating the cell grid.
	//
	// cell index
	// | 0 | 1 | 2 | 3 | 4 |   
	//
	// ptc_to_cell
	// - size: #particles
	// - a map between each particle index to its cell 
	//   (NOT the active cell) index
	//
	// cell_num_ptcs
	//   3   0   1   4   0
	// - size: #cells
	// - number of particles in each cell, including inactive(empty) ones
	// - sum of this array is the total number of particles
	//
	// cell_is_active_flags
	//   1   0   1   1   0
	// - size: #cells
	// - sum of this array is the number of active cells
	//
	// cell_to_active_cell_indices
	//   0   1   1   2   3
	// - size: #cells
	// - a prefix scan of |cell_is_active_flags|
	//
	// active_cell_num_ptcs
	//   3   1   4
	// - size: #active cells
	// - a compact of |cell_num_ptcs| accoording to |cell_to_active_cell_indices|
	// - sum of this array is the total number of particles
	//
	// ptc_begins_in_active_cell 
	//   0   3   4
	// - size: #active cells
	// - beginning index of the particle in each (active) cell in 
	//   |cell_ptc_indices|
	// - a prefix scan of |active_cell_num_ptcs|
	//
	// cell_ptc_indices
	// - size: #ptcs
	// - each slot stores a particle index in the particle system. these particle
	//   indices are arranged in a way that particles within the same grid cell
	//   are continuously stored inside |cell_ptc_indices|.
	//
	// ptc_offsets_within_cell
	// - size: #particles
	// - for any given particle index, |p_i|, we can get its position, |pos_i|,
	//   and its cell, |cell_i|. Then:
	//   // the active cell index of |cell_i| in which |ptc_i| lives
	//   ac_idx = cell_to_active_cell_indices[cell_i];
	//   // the beginning index of the particles within |cell_i|
	//   // in |cell_ptc_indices|
	//   ptc_begin_idx = ptc_begins_in_active_cell[ac_idx];
	//   p_i' = cell_ptc_indices[ptc_begin_idx + ptc_offset_within_cell[p_i]]; 
	//   assert(p_i == p_i');
	//
	// Find neighbors for each particle:
	// ptc_num_neighbors
	// - size: #particles
	// - stores the number of neighbor particles for each particle
	//
	// ptc_neighbor_begins
	// - size: #particles
	// - ~[p_i] stores the beginning index of particle |p_i|'s neighbors
	//   in |ptc_neighbor_indices|
	//
	// ptc_neighbor_indices
	// - size: sum of |ptc_num_neighbors| 

	int ComputeNumBlocks(int num) {
		return ((num + kNumThreadPerBlock - 1) / kNumThreadPerBlock);
	}

	__device__ int3 GetCell(float3 pos, float cell_sz) {
		const float cell_sz_recpr = 1.0f / cell_sz;
		int cx = (int)(pos.x * cell_sz_recpr);
		int cy = (int)(pos.y * cell_sz_recpr);
		int cz = (int)(pos.z * cell_sz_recpr);
		return make_int3(cx, cy, cz);
	}

	__device__ int GetCellIndex(int3 cell, int3 num_cells_dim) {
		int result = cell.y * num_cells_dim.z;
		result = (result + cell.z) * num_cells_dim.x;
		result += cell.x;
		return result;
	}

	__device__ bool CellOutOfRange(int3 cell, int3 num_cells_dim) {
		return ((0 <= cell.x && cell.x < num_cells_dim.x) &&
			(0 <= cell.y && cell.y < num_cells_dim.y) &&
			(0 <= cell.z && cell.z < num_cells_dim.z));
	}

	__device__ float DistanceSquare(float3 a, float3 b) {
		float x = a.x - b.x;
		float y = a.y - b.y;
		float z = a.z - b.z;
		float result = x * x + y * y + z * z;
		return result;
	}
	/////
	// CellGrid
	/////
	void ResetNumPtcsInCell(d_vector<int>* cell_num_ptcs) {
		const size_t sz = cell_num_ptcs->size();
		cell_num_ptcs->assign(sz, 0);
	}

	// - compute |ptc_to_cell| 
	// - count |cell_num_ptcs|
	// - set the offset of each partilce in |ptc_offset_within_cell|.
	__global__ void CellGridEntryPointKernel(const float3* positions,
		const int num_ptcs, const float cell_sz, const int3 num_cells_dim,
		int* ptc_to_cell, int* cell_num_ptcs , int* ptc_offsets_within_cell) 
	{
		const int ptc_i = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (ptc_i >= num_ptcs) return;
		
		int3 ptc_cell = GetCell(positions[ptc_i], cell_sz);
		int cell_index = GetCellIndex(ptc_cell, num_cells_dim);
		
		ptc_to_cell[ptc_i] = cell_index;
		// Count the number of particles in |ptc_cell|. The returned
		// value is also used as this particle's unique offset.
		int offs = atomicAdd(&cell_num_ptcs[cell_index], 1);
		(void)offs;
		ptc_offsets_within_cell[ptc_i] = offs;
	}

	// set |cell_is_active_flags|
	__global__ void SetCellIsActiveFlagsKernel(const int* cell_num_ptcs,
		const int num_cells, int* cell_is_active_flags) 
	{
		const int cell_i = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (cell_i >= num_cells) return;
		cell_is_active_flags[cell_i] = (cell_num_ptcs[cell_i] > 0);
	}

	// compute |cell_to_active_cell_indices|
	void ComputeCellToActiveCellIndices(
		const d_vector<int>& cell_is_active_flags,
		d_vector<int>* cell_to_active_cell_indices) 
	{
		assert(cell_is_active_flags.size() == 
			cell_to_active_cell_indices->size());
		thrust::exclusive_scan(thrust::device, 
			cell_is_active_flags.begin(), cell_is_active_flags.end(),
			cell_to_active_cell_indices->begin(), 0);
	}

	__global__ void CompactKernel(const int* input, const int* flag, 
		const int* compact_indices, const int size, int* output) 
	{
		const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (idx >= size) return;
		if (flag[idx] != 0) {
			const int compact_idx = compact_indices[idx];
			output[compact_idx] = input[idx];
		}
	}

	// compact |cell_num_ptcs| to get |active_cell_num_ptcs|
	void ComputeActiveCellNumPtcs(const d_vector<int>& cell_num_ptcs,
		const d_vector<int>& cell_is_active_flags,
		const d_vector<int>& cell_to_active_cell_indices,
		d_vector<int>* active_cell_num_ptcs)
	{
		const int size = cell_is_active_flags.size();
		const int num_blocks = ComputeNumBlocks(size);
		const int* input = thrust::raw_pointer_cast(cell_num_ptcs.data());
		const int* flags = thrust::raw_pointer_cast(cell_is_active_flags.data());
		const int* compact_indices = thrust::raw_pointer_cast(
			cell_to_active_cell_indices.data());
		int* output = thrust::raw_pointer_cast(active_cell_num_ptcs->data());
		CompactKernel<<<num_blocks, kNumThreadPerBlock>>> (
			input, flags, compact_indices, size, output);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
	}

	// compute |ptc_begins_in_active_cell|
	void ComputePtcBeginsInActiveCell(
		const d_vector<int>& active_cell_num_ptcs,
		d_vector<int>* ptc_begins_in_active_cell)
	{
		assert(active_cell_num_ptcs.size() == 
			ptc_begins_in_active_cell->size());
		thrust::exclusive_scan(thrust::device, 
			active_cell_num_ptcs.begin(), active_cell_num_ptcs.end(),
			ptc_begins_in_active_cell->begin(), 0);
	}

	// compute |cell_ptc_indices|
	__global__ void ComputeCellPtcIndicesKernel(
		const int* ptc_to_cell, const int* cell_to_active_cell_indices, 
		const int* ptc_begins_in_active_cell, 
		const int* ptc_offsets_within_cell,
		const int num_ptcs, int* cell_ptc_indices)
	{
		const int ptc_i = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (ptc_i >= num_ptcs) return;
		
		const int cell_i = ptc_to_cell[ptc_i];
		// active cell index
		const int ac_idx = cell_to_active_cell_indices[cell_i];
		const int ptc_begin_index = ptc_begins_in_active_cell[ac_idx];
		const int i = ptc_begin_index + ptc_offsets_within_cell[ptc_i];
		cell_ptc_indices[i] = ptc_i;
	}

	/////
	// Find Neighbor Particles
	/////

	// Count |ptc_num_neighbors|
	// - |radius|: searching radius
	__global__ void CountPtcNumNeighborsKernel(const float3* positions, 
		const int* cell_to_active_cell_indices, const int* cell_ptc_indices, 
		const int* ptc_begins_in_active_cell, const int* active_cell_num_ptcs,
		const int num_ptcs, const float cell_sz, const int3 num_cells_dim,
		const float radius, int* ptc_num_neighbors)
	{
		const int ptc_i = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (ptc_i >= num_ptcs) return;

		int3 ptc_cell = GetCell(positions[ptc_i], cell_sz);
		int num_neighbors = 0;
		const float radius_sqr = radius * radius;
		const float3 pos_i = positions[ptc_i];
		// We are only checking the 8 adjacent cells plus the cell itself,
		// this implies that our cell size must be greater than |radius|.
		for (int cz = -1; cz <= 1; ++cz) {
			for (int cy = -1; cy <= 1; ++cy) {
				for (int cx = -1; cx <= 1; ++cx) {
					int3 nb_cell = ptc_cell + make_int3(cx, cy, cz);
					if (CellOutOfRange(nb_cell, num_cells_dim))
						continue;
					int nb_cell_idx = GetCellIndex(nb_cell, num_cells_dim);
					const int nb_ac_idx = 
						cell_to_active_cell_indices[nb_cell_idx];
					const int ac_num_ptcs = active_cell_num_ptcs[nb_ac_idx];
					const int nb_ptc_begin = ptc_begins_in_active_cell[nb_ac_idx];
					for (int offs = 0; offs < ac_num_ptcs; ++offs) {
						const int ptc_j = cell_ptc_indices[nb_ptc_begin + offs];
						float dist_sqr = DistanceSquare(pos_i, positions[ptc_j]);
						if (dist_sqr < radius_sqr) {
							++num_neighbors;
						}
					}
				}
			}
		}
		ptc_num_neighbors[ptc_i] = num_neighbors;
	}
	
	// compute |ptc_neighbor_begins|
	void ComputePtcNeighborBegins(const d_vector<int>& ptc_num_neighbors,
		d_vector<int>* ptc_neighbor_begins) 
	{
		assert(ptc_num_neighbors.size() == 
			ptc_neighbor_begins->size());
		thrust::exclusive_scan(thrust::device, 
			ptc_num_neighbors.begin(), ptc_num_neighbors.end(),
			ptc_neighbor_begins->begin(), 0);
	}

	// Find neighbor particles and store them in |ptc_neighbor_indices|
	// - |radius|: searching radius
	__global__ void FindPtcNeighborIndicesKernel(const float3* positions, 
		const int* cell_to_active_cell_indices, const int* cell_ptc_indices, 
		const int* ptc_begins_in_active_cell, const int* active_cell_num_ptcs,
		const int num_ptcs, const float cell_sz, const int3 num_cells_dim,
		const float radius, int* ptc_neighbor_begins, int* ptc_neighbor_indices, 
		const int* ptc_num_neighbors /*debug purpose, rm once correct*/)
	{
		const int ptc_i = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (ptc_i >= num_ptcs) return;

		int3 ptc_cell = GetCell(positions[ptc_i], cell_sz);
		int cur = ptc_neighbor_begins[ptc_i];
		const int cur_copy = cur;
		const float radius_sqr = radius * radius;
		const float3 pos_i = positions[ptc_i];
		// We are only checking the 8 adjacent cells plus the cell itself,
		// this implies that our cell size must be greater than |radius|.
		for (int cz = -1; cz <= 1; ++cz) {
			for (int cy = -1; cy <= 1; ++cy) {
				for (int cx = -1; cx <= 1; ++cx) {
					int3 nb_cell = ptc_cell + make_int3(cx, cy, cz);
					if (CellOutOfRange(nb_cell, num_cells_dim))
						continue;
					int nb_cell_idx = GetCellIndex(nb_cell, num_cells_dim);
					const int nb_ac_idx = 
						cell_to_active_cell_indices[nb_cell_idx];
					const int ac_num_ptcs = active_cell_num_ptcs[nb_ac_idx];
					const int nb_ptc_begin = ptc_begins_in_active_cell[nb_ac_idx];
					for (int offs = 0; offs < ac_num_ptcs; ++offs) {
						const int ptc_j = cell_ptc_indices[nb_ptc_begin + offs];
						float dist_sqr = DistanceSquare(pos_i, positions[ptc_j]);
						if (dist_sqr < radius_sqr) {
							ptc_neighbor_indices[cur] = ptc_j;
							++cur;
						}
					}
				}
			}
		}
		// Use GPU assert!
		// assert((cur - cur_copy) == ptc_num_neighbors[ptc_i]);
	}
} // namespace impl_
		
	CellGridGpu::CellGridGpu(float3 world_sz, float cell_sz)
		: world_sz_per_dim_(world_sz), cell_sz_(cell_sz) {
		num_cells_per_dim_.x = (int)(world_sz_per_dim_.x / cell_sz_) + 1;
		num_cells_per_dim_.y = (int)(world_sz_per_dim_.y / cell_sz_) + 1;
		num_cells_per_dim_.z = (int)(world_sz_per_dim_.z / cell_sz_) + 1;

		total_num_cells_ = num_cells_per_dim_.x * num_cells_per_dim_.y
			* num_cells_per_dim_.z;
	}

	void UpdateCellGrid(const d_vector<float3>& positions, CellGridGpu* cell_grid)
	{
		using thrust::raw_pointer_cast;
		using namespace impl_;

		const int num_ptcs = positions.size();
		const int num_cells = cell_grid->total_num_cells();

		const float3* positions_ptr = raw_pointer_cast(positions.data());
		// d_vector<int> ptc_to_cell(num_ptcs, 0);
		d_vector<int>& ptc_to_cell = cell_grid->ptc_to_cell;
		ptc_to_cell.clear();
		ptc_to_cell.resize(num_ptcs, 0);
		int* ptc_to_cell_ptr = raw_pointer_cast(ptc_to_cell.data());
		d_vector<int> cell_num_ptcs(num_cells, 0);
		int* cell_num_ptcs_ptr = raw_pointer_cast(cell_num_ptcs.data());
		// d_vector<int> ptc_offsets_within_cell(num_ptcs, 0);
		d_vector<int>& ptc_offsets_within_cell = 
			cell_grid->ptc_offsets_within_cell;
		ptc_offsets_within_cell.clear();
		ptc_offsets_within_cell.resize(num_ptcs, 0);
		int* ptc_offsets_within_cell_ptr = 
			raw_pointer_cast(ptc_offsets_within_cell.data());

		const int num_blocks_ptc = ComputeNumBlocks(num_ptcs);
		CellGridEntryPointKernel<<<num_blocks_ptc, kNumThreadPerBlock>>>(
			positions_ptr, num_ptcs, cell_grid->cell_size(), 
			cell_grid->num_cells_per_dim(), ptc_to_cell_ptr, 
			cell_num_ptcs_ptr, ptc_offsets_within_cell_ptr);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());

		// d_vector<int> cell_is_active_flags(num_cells, 0);
		d_vector<int>& cell_is_active_flags = cell_grid->cell_is_active_flags;
		cell_is_active_flags.clear();
		cell_is_active_flags.resize(num_cells, 0);
		int* cell_is_active_flags_ptr = raw_pointer_cast(
			cell_is_active_flags.data());
		
		const int num_blocks_cell = ComputeNumBlocks(num_cells);
		SetCellIsActiveFlagsKernel<<<num_blocks_cell, kNumThreadPerBlock>>>(
			cell_num_ptcs_ptr, num_cells, cell_is_active_flags_ptr);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());

		d_vector<int>& cell_to_active_cell_indices = 
			cell_grid->cell_to_active_cell_indices;
		cell_to_active_cell_indices.clear();
		cell_to_active_cell_indices.resize(num_cells, 0);
		ComputeCellToActiveCellIndices(cell_is_active_flags,
			&cell_to_active_cell_indices);

		d_vector<int>& active_cell_num_ptcs = cell_grid->active_cell_num_ptcs;
		active_cell_num_ptcs.clear();
		active_cell_num_ptcs.resize(num_cells, 0);
		ComputeActiveCellNumPtcs(cell_num_ptcs, cell_is_active_flags,
			cell_to_active_cell_indices, &active_cell_num_ptcs);

		d_vector<int>& ptc_begins_in_active_cell =
			cell_grid->ptc_begins_in_active_cell;
		ptc_begins_in_active_cell.clear();
		ptc_begins_in_active_cell.resize(num_cells, 0);
		ComputePtcBeginsInActiveCell(
			active_cell_num_ptcs, &ptc_begins_in_active_cell);

		const int* cell_to_active_cell_indices_ptr =
			raw_pointer_cast(cell_to_active_cell_indices.data());
		const int* ptc_begins_in_active_cell_ptr =
			raw_pointer_cast(ptc_begins_in_active_cell.data());
		d_vector<int>& cell_ptc_indices = cell_grid->cell_ptc_indices;
		cell_ptc_indices.clear();
		cell_ptc_indices.resize(num_ptcs, 0);
		int* cell_ptc_indices_ptr = raw_pointer_cast(cell_ptc_indices.data());
		ComputeCellPtcIndicesKernel<<<num_blocks_ptc, kNumThreadPerBlock>>>(
			ptc_to_cell_ptr, cell_to_active_cell_indices_ptr,
			ptc_begins_in_active_cell_ptr, ptc_offsets_within_cell_ptr,
			num_ptcs, cell_ptc_indices_ptr);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
	}

	__device__ bool IsInside(const float3& pt, const 
		float3& min, const float3& max) {
        bool cond = (min.x <= pt.x) && (pt.x <= max.x) &&
        (min.y <= pt.y) && (pt.y <= max.y) &&
        (min.z <= pt.z) && (pt.z <= max.z);
        return cond;
	}
	
	__global__ static void QueryCountKernel(const int num_cells, 
		const float3 range_min, const float3 range_max, const float3* positions, 
		const int* cell_is_active_flags, const int* cell_to_active_cell_indices,
		const int* ptc_begins_in_active_cell, const int* active_cell_num_ptcs,
		const int* cell_ptc_indices, int* cell_num_ptcs_inside) {
		int cell_i = (blockDim.x * blockIdx.x) + threadIdx.x;
		if (cell_i >= num_cells) return;
		
		bool is_active = cell_is_active_flags[cell_i];;
		if (!is_active) return;

		const int ac_idx = cell_to_active_cell_indices[cell_i];
		const int ptc_begin = ptc_begins_in_active_cell[ac_idx];
		const int ac_num_ptcs = active_cell_num_ptcs[ac_idx];
		int num_inside = 0;
		for (int offs = 0; offs < ac_num_ptcs; ++offs) {
			int ptc_i = cell_ptc_indices[ptc_begin + offs];
			if (IsInside(positions[ptc_i], range_min, range_max)) {
				++num_inside;
			}
		}
		cell_num_ptcs_inside[cell_i] = num_inside;
	}
	
	void Query(const d_vector<float3>& positions, const CellGridGpu& cell_grid, 
		const AABB& range, d_vector<int>* cell_num_ptcs_inside) {
		using namespace impl_;
		using thrust::raw_pointer_cast;

		const int num_cells = cell_grid.total_num_cells();
		const float3 range_min = Convert(range.min());
		const float3 range_max = Convert(range.max());
		const float3* positions_ptr = raw_pointer_cast(positions.data());
		const int* cell_is_active_flags_ptr =
			raw_pointer_cast(cell_grid.cell_is_active_flags.data());
		const int* cell_to_active_cell_indices_ptr =
			raw_pointer_cast(cell_grid.cell_to_active_cell_indices.data());
		const int* ptc_begins_in_active_cell_ptr =
			raw_pointer_cast(cell_grid.ptc_begins_in_active_cell.data());
		const int* active_cell_num_ptcs_ptr =
			raw_pointer_cast(cell_grid.active_cell_num_ptcs.data());
		const int* cell_ptc_indices_ptr = 
			raw_pointer_cast(cell_grid.cell_ptc_indices.data());
		cell_num_ptcs_inside->clear();
		cell_num_ptcs_inside->resize(num_cells, 0);
		int* cell_num_ptcs_inside_ptr =
			raw_pointer_cast(cell_num_ptcs_inside->data());

		const int num_blocks_cell = ComputeNumBlocks(num_cells);
		QueryCountKernel<<<num_blocks_cell, kNumThreadPerBlock>>>(
			num_cells, range_min, range_max, positions_ptr, 
			cell_is_active_flags_ptr, cell_to_active_cell_indices_ptr,
			ptc_begins_in_active_cell_ptr, active_cell_num_ptcs_ptr,
			cell_ptc_indices_ptr, cell_num_ptcs_inside_ptr);
	}
} // namespace pbf