#include "hip/hip_runtime.h"
#include "../include/pbf_solver_gpu.h"

// CUDA
#include "hip/hip_runtime.h"
#include <thrust\device_vector.h>
#include <thrust\scan.h>
#include <thrust\execution_policy.h>
#include <thrust\copy.h>

namespace pbf {
namespace impl_ {
	constexpr int kNumThreadPerBlock = 256;
	
	template <typename T>
	using d_vector = thrust::device_vector<T>;
	// ParticleSystemGpu
	//
	// CellGridGpu (equivalent to SpatialHash on CPU)
	// - Need to config the grid size and cell size before usage. Once
	//   configured, they are not allowed to be modified. Not adaptive.
	// - active cell: a cell that contains at least one particle.
	//
	// Example: a particle system of 8 particles, a cell grid of 5 cells.
	// We will illustrate the necessary arrays for updating the cell grid.
	//
	// cell index
	// | 0 | 1 | 2 | 3 | 4 |   
	//
	// cell_num_ptcs
	//   3   0   1   4   0
	// - size: #cells
	// - number of particles in each cell, including inactive(empty) ones
	// - sum of this array is the total number of particles
	//
	// cell_is_active_flags
	//   1   0   1   1   0
	// - size: #cells
	// - sum of this array is the number of active cells
	//
	// cell_to_active_cell_indices
	//   0   1   1   2   3
	// - size: #cells
	// - a prefix scan of |cell_is_active_flags|
	//
	// active_cell_num_ptcs
	//   3   1   4
	// - size: #active cells
	// - a compact of |cell_num_ptcs| accoording to |cell_to_active_cell_indices|
	// - sum of this array is the total number of particles
	//
	// ptc_begins_in_active_cell 
	//   0   3   4
	// - size: #active cells
	// - beginning index of the particle in each (active) cell in 
	//   |cell_ptc_indices|
	// - a prefix scan of |active_cell_num_ptcs|
	//
	// cell_ptc_indices
	// - size: #ptcs
	// - each slot stores a particle index in the particle system. these particle
	//   indices are arranged in a way that particles within the same grid cell
	//   are continuously stored inside |cell_ptc_indices|.
	//
	// ptc_offsets_within_cell
	// - size: #particles
	// - for any given particle index, |p_i|, we can get its position, |pos_i|,
	//   and its cell, |cell_i|. Then:
	//	 // the active cell index of |cell_i| in which |ptc_i| lives
	//	 ac_idx = cell_to_active_cell_indices[cell_i];
	//   // the beginning index of the particles within |cell_i|
	//	 // in |cell_ptc_indices|
	//   ptc_begin_idx = ptc_begins_in_active_cell[ac_index];
	//   p_i' = cell_ptc_indices[ptc_begin_idx + ptc_offset_within_cell[p_i]]; 
	//   assert(p_i == p_i');
	//
	// Find neighbors for each particle:
	// ptc_num_neighbors
	// - size: #particles
	// - stores the number of neighbor particles for each particle
	//
	// ptc_neighbor_begins
	// - size: #particles
	// - ~[p_i] stores the beginning index of particle |p_i|'s neighbors
	//   in |ptc_neighbor_indices|
	//
	// ptc_neighbor_indices
	// - size: sum of |ptc_num_neighbors| 
	
	__device__ int3 GetCell(float3 pos, float cell_sz) {
		const float cell_sz_recpr = 1.0f / cell_sz;
		int cx = (int)(pos.x * cell_sz_recpr);
		int cy = (int)(pos.y * cell_sz_recpr);
		int cz = (int)(pos.z * cell_sz_recpr);
		return make_int3(cx, cy, cz);
	}

	__device__ int GetCellIndex(int3 cell, int3 num_cells_dim) {
		int result = cell.y * num_cells_dim.z;
		result = (result + cell.z) * num_cells_dim.x;
		result += cell.x;
		return result;
	}

	__device__ bool IsValidCell(int3 cell, int3 num_cells_dim) {
		return ((0 <= cell.x && cell.x < num_cells_dim.x) &&
			(0 <= cell.y && cell.y < num_cells_dim.y) &&
			(0 <= cell.z && cell.z < num_cells_dim.z));
	}

	void ResetNumPtcsInCell(d_vector<int>* cell_num_ptcs) {
		const size_t sz = cell_num_ptcs->size();
		cell_num_ptcs->assign(sz, 0);
	}

	// count |cell_num_ptcs| and set the offset of each partilce
	// in |ptc_offset_within_cell|.
	__global__ void CountPtcsAndSetPtcOffsetsInCell(
		const float3* positions, const int num_ptcs, 
		const float cell_sz, const int3 num_cells_dim, 
		int* cell_num_ptcs, int* ptc_offset_within_cell) 
	{
		const int ptc_i = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (ptc_i >= num_ptcs) return;
		int3 ptc_cell = GetCell(positions[ptc_i], cell_sz);
		int cell_index = GetCellIndex(ptc_cell, num_cells_dim);
		// Count the number of particles in |ptc_cell|. The returned
		// value is also used as this particle's unique offset.
		int offs = atomicAdd(&cell_num_ptcs[cell_index], 1);
		ptc_offset_within_cell[ptc_i] = offs;
	}

	// set |cell_is_active_flags|
	__global__ void SetCellIsActiveFlags(const int* cell_num_ptcs,
		const int num_cells, int* cell_is_active_flags) 
	{
		const int cell_i = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (cell_i >= num_cells) return;
		cell_is_active_flags[cell_i] = (cell_num_ptcs[cell_i] > 0);
	}

	// compute |cell_to_active_cell_indices|
	void ComputeCellToActiveCellIndices(
		const d_vector<int>& cell_is_active_flags,
		d_vector<int>* cell_to_active_cell_indices) 
	{
		assert(cell_is_active_flags.size() == 
			cell_to_active_cell_indices->size());
		thrust::exclusive_scan(thrust::device, 
			cell_is_active_flags.begin(), cell_is_active_flags.end(),
			cell_to_active_cell_indices->begin(), 0);
	}

	__global__ void Compact(const int* input, const int* flag, 
		const int* compact_indices, const int size, int* output) 
	{
		const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (idx >= size) return;
		if (flag[idx] != 0) {
			const int compact_idx = compact_indices[idx];
			output[compact_idx] = input[idx];
		}
	}

	// compact |cell_num_ptcs| to get |active_cell_num_ptcs|
	void ComputeActiveCellNumPtcs(const d_vector<int>& cell_num_ptcs,
		const d_vector<int>& cell_is_active_flags,
		const d_vector<int>& cell_to_active_cell_indices,
		d_vector<int>* active_cell_num_ptcs)
	{
		const int size = cell_is_active_flags.size();
		const int num_blocks = ((size + kNumThreadPerBlock - 1) /
			kNumThreadPerBlock);
		const int* input = thrust::raw_pointer_cast(cell_num_ptcs.data());
		const int* flags = thrust::raw_pointer_cast(cell_is_active_flags.data());
		const int* compact_indices = thrust::raw_pointer_cast(
			cell_to_active_cell_indices.data());
		int* output = thrust::raw_pointer_cast(active_cell_num_ptcs->data());
		Compact <<<num_blocks, kNumThreadPerBlock>>> (
			input, flags, compact_indices, size, output);
	}

	// compute |ptc_begins_in_active_cell|
	void ComputePtcBeginsInActiveCell(
		const d_vector<int>& active_cell_num_ptcs,
		d_vector<int>* ptc_begins_in_active_cell)
	{
		assert(active_cell_num_ptcs.size() == 
			ptc_begins_in_active_cell->size());
		thrust::exclusive_scan(thrust::device, 
			active_cell_num_ptcs.begin(), active_cell_num_ptcs.end(),
			ptc_begins_in_active_cell->begin(), 0);
	}

	// compute |cell_ptc_indices|
	__global__ void ComputeCellPtcIndices(
		const int num_ptcs, int* cell_ptc_indices)
	{

	}
} // namespace impl_
} // namespace pbf